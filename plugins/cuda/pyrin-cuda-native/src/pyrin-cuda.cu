#include "hip/hip_runtime.h"
// CUDA Pyrin Hash Optimized Kernel

#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>

// Assuming keccak and blake3 implementations have been included properly
#include "keccak-tiny.h"
#include "blake3_compact.h"

__device__ uint64_t xoshiro256starstar(uint64_t* state) {
    uint64_t result = rotl(state[1] * 5, 7) * 9;
    uint64_t t = state[1] << 17;

    state[2] ^= state[0];
    state[3] ^= state[1];
    state[1] ^= state[2];
    state[0] ^= state[3];

    state[2] ^= t;
    state[3] = rotl(state[3], 45);

    return result;
}

__global__ void pyrin_cuda_kernel(uint8_t* input_data, uint8_t* output_data, size_t data_size) {
    extern __shared__ uint8_t shared_data[];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= data_size) return;

    // Load input data to shared memory
    shared_data[threadIdx.x] = input_data[idx];
    __syncthreads();

    // Perform keccak hash on the input
    uint8_t keccak_output[32];
    keccak(shared_data, blockDim.x, keccak_output, sizeof(keccak_output));

    // Perform blake3 hash on keccak output
    uint8_t blake3_output[32];
    blake3_hasher hasher;
    blake3_hasher_init(&hasher);
    blake3_hasher_update(&hasher, keccak_output, sizeof(keccak_output));
    blake3_hasher_finalize(&hasher, blake3_output, sizeof(blake3_output));

    // Write the final output back to global memory
    output_data[idx] = blake3_output[0];
}

extern "C" void launch_pyrin_cuda_kernel(uint8_t* input_data, uint8_t* output_data, size_t data_size) {
    int block_size = 256;
    int grid_size = (data_size + block_size - 1) / block_size;
    int shared_memory_size = block_size * sizeof(uint8_t);

    pyrin_cuda_kernel<<<grid_size, block_size, shared_memory_size>>>(input_data, output_data, data_size);
    hipDeviceSynchronize();
}
